// hough_transform.cu

#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <limits.h>

extern "C" {

// 定义常量
#define NUM_THETA 360

__constant__ float d_cos_theta[NUM_THETA];
__constant__ float d_sin_theta[NUM_THETA];

// CUDA核函数：计算Hough累加器
__global__ void houghTransformKernel(
    const unsigned char* edge_image,
    int rows, int cols,
    int r,
    int hough_max_x, int hough_max_y,
    int* accumulator)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = rows * cols;

    if (idx >= total_pixels)
        return;

    int y = idx / cols;
    int x = idx % cols;

    if (edge_image[y * cols + x] > 0)
    {
        // 对每个theta值进行计算
        for (int theta_idx = 0; theta_idx < NUM_THETA; ++theta_idx)
        {
            float cos_t = d_cos_theta[theta_idx];
            float sin_t = d_sin_theta[theta_idx];

            float a = x - r * cos_t;
            float b = y - r * sin_t;

            int a_idx = (int)roundf(a + r);
            int b_idx = (int)roundf(b + r);

            if (a_idx >= 0 && a_idx < hough_max_x && b_idx >= 0 && b_idx < hough_max_y)
            {
                atomicAdd(&accumulator[b_idx * hough_max_x + a_idx], 1);
            }
        }
    }
}

// CUDA核函数：找到累加器中的最大值
__global__ void findAccumulatorMaxKernel(
    const int* accumulator,
    int accumulator_size,
    int* max_value)
{
    extern __shared__ int shared_max[];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int tid = threadIdx.x;

    // 初始化共享内存
    shared_max[tid] = 0;

    // 加载数据到共享内存
    if (idx < accumulator_size)
    {
        shared_max[tid] = accumulator[idx];
    }

    __syncthreads();

    // 归约求最大值
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride && (idx + stride) < accumulator_size)
        {
            if (shared_max[tid] < shared_max[tid + stride])
            {
                shared_max[tid] = shared_max[tid + stride];
            }
        }
        __syncthreads();
    }

    // 将每个块的最大值写入全局内存
    if (tid == 0)
    {
        atomicMax(max_value, shared_max[0]);
    }
}

// CUDA核函数：收集最大值的位置
__global__ void collectMaxPositionsKernel(
    const int* accumulator,
    int hough_max_x, int hough_max_y,
    int max_value,
    int* max_positions, // 存储位置的数组 [a1, b1, a2, b2, ...]
    int* max_count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_size = hough_max_x * hough_max_y;

    if (idx >= total_size)
        return;

    int value = accumulator[idx];

    if (value == max_value)
    {
        int pos = atomicAdd(max_count, 1);
        max_positions[2 * pos] = idx % hough_max_x;
        max_positions[2 * pos + 1] = idx / hough_max_x;
    }
}

// CUDA核函数：计算9x9邻域的均值
__global__ void computeNeighborhoodAveragesKernel(
    const int* accumulator,
    int hough_max_x, int hough_max_y,
    int* max_positions,
    int max_count,
    float* averages)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= max_count)
        return;

    int a = max_positions[2 * idx];
    int b = max_positions[2 * idx + 1];

    int sum = 0;
    int count = 0;

    for (int dy = -3; dy <= 3; ++dy)
    {
        for (int dx = -3; dx <= 3; ++dx)
        {
            int x = a + dx;
            int y = b + dy;
            if (x >= 0 && x < hough_max_x && y >= 0 && y < hough_max_y)
            {
                sum += accumulator[y * hough_max_x + x];
                count++;
            }
        }
    }

    averages[idx] = (float)sum / count;
}

// 主机函数，供Python调用
__host__ void hough_transform_for_radius(
    const unsigned char* edge_image,
    int rows, int cols,
    int r,
    int* best_a, int* best_b,
    int* accumulator_out)
{
    int hough_max_x = 2 * r + cols;
    int hough_max_y = 2 * r + rows;

    int accumulator_size = hough_max_x * hough_max_y;

    // 分配设备内存
    unsigned char* d_edge_image;
    int* d_accumulator;
    hipError_t err;

    // 检查CUDA错误的宏定义
    #define CHECK_CUDA_ERROR(call) \
        do { \
            hipError_t err = call; \
            if (err != hipSuccess) { \
                fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
                return; \
            } \
        } while (0)

    // 分配d_edge_image
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_edge_image, rows * cols * sizeof(unsigned char)));
    CHECK_CUDA_ERROR(hipMemcpy(d_edge_image, edge_image, rows * cols * sizeof(unsigned char), hipMemcpyHostToDevice));

    // 分配d_accumulator
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_accumulator, accumulator_size * sizeof(int)));
    CHECK_CUDA_ERROR(hipMemset(d_accumulator, 0, accumulator_size * sizeof(int)));

    // 初始化cos和sin数组
    float h_theta[NUM_THETA];
    for (int i = 0; i < NUM_THETA; ++i)
    {
        float theta = i * 2.0f * M_PI / NUM_THETA;
        h_theta[i] = theta;
    }

    float h_cos_theta[NUM_THETA];
    float h_sin_theta[NUM_THETA];

    for (int i = 0; i < NUM_THETA; ++i)
    {
        h_cos_theta[i] = cosf(h_theta[i]);
        h_sin_theta[i] = sinf(h_theta[i]);
    }

    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_cos_theta), h_cos_theta, NUM_THETA * sizeof(float)));
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_sin_theta), h_sin_theta, NUM_THETA * sizeof(float)));

    // 启动Hough变换核函数
    int total_pixels = rows * cols;
    int threadsPerBlock = 256;
    int blocksPerGrid = (total_pixels + threadsPerBlock - 1) / threadsPerBlock;

    houghTransformKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_edge_image, rows, cols, r, hough_max_x, hough_max_y, d_accumulator);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // 找到累加器中的最大值
    int* d_max_value;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_max_value, sizeof(int)));
    CHECK_CUDA_ERROR(hipMemset(d_max_value, 0, sizeof(int)));

    int accumulator_blocks = (accumulator_size + threadsPerBlock - 1) / threadsPerBlock;

    size_t sharedMemSize = threadsPerBlock * sizeof(int);

    findAccumulatorMaxKernel<<<accumulator_blocks, threadsPerBlock, sharedMemSize>>>(
        d_accumulator, accumulator_size, d_max_value);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    int h_max_value = 0;
    CHECK_CUDA_ERROR(hipMemcpy(&h_max_value, d_max_value, sizeof(int), hipMemcpyDeviceToHost));

    // 收集最大值的位置
    int* d_max_positions;
    int* d_max_count;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_max_positions, accumulator_size * 2 * sizeof(int))); // 最坏情况下，所有位置都是最大值
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_max_count, sizeof(int)));
    CHECK_CUDA_ERROR(hipMemset(d_max_count, 0, sizeof(int)));

    collectMaxPositionsKernel<<<accumulator_blocks, threadsPerBlock>>>(
        d_accumulator, hough_max_x, hough_max_y, h_max_value, d_max_positions, d_max_count);
    CHECK_CUDA_ERROR(hipGetLastError());
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    int h_max_count = 0;
    CHECK_CUDA_ERROR(hipMemcpy(&h_max_count, d_max_count, sizeof(int), hipMemcpyDeviceToHost));

    printf("Number of positions with max value: %d\n", h_max_count);

    if (h_max_count == 0)
    {
        fprintf(stderr, "No positions with max value found.\n");
        *best_a = -1;
        *best_b = -1;
    }
    else if (h_max_count == 1)
    {
        int h_max_positions[2];
        CHECK_CUDA_ERROR(hipMemcpy(h_max_positions, d_max_positions, 2 * sizeof(int), hipMemcpyDeviceToHost));
        *best_a = h_max_positions[0];
        *best_b = h_max_positions[1];
    }
    else
    {
        // 计算每个最大值位置的邻域均值
        float* d_averages;
        CHECK_CUDA_ERROR(hipMalloc((void**)&d_averages, h_max_count * sizeof(float)));

        int avg_blocks = (h_max_count + threadsPerBlock - 1) / threadsPerBlock;

        computeNeighborhoodAveragesKernel<<<avg_blocks, threadsPerBlock>>>(
            d_accumulator, hough_max_x, hough_max_y, d_max_positions, h_max_count, d_averages);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // 将均值复制回主机
        float* h_averages = new float[h_max_count];
        CHECK_CUDA_ERROR(hipMemcpy(h_averages, d_averages, h_max_count * sizeof(float), hipMemcpyDeviceToHost));

        // 将位置复制回主机
        int* h_max_positions = new int[h_max_count * 2];
        CHECK_CUDA_ERROR(hipMemcpy(h_max_positions, d_max_positions, h_max_count * 2 * sizeof(int), hipMemcpyDeviceToHost));

        // 找到均值最大的那个位置
        int best_idx = 0;
        float max_average = h_averages[0];
        for (int i = 1; i < h_max_count; ++i)
        {
            if (h_averages[i] > max_average)
            {
                max_average = h_averages[i];
                best_idx = i;
            }
        }

        *best_a = h_max_positions[2 * best_idx];
        *best_b = h_max_positions[2 * best_idx + 1];

        // 释放主机内存
        delete[] h_averages;
        delete[] h_max_positions;
        CHECK_CUDA_ERROR(hipFree(d_averages));
    }

    // 将累加器复制回主机
    CHECK_CUDA_ERROR(hipMemcpy(accumulator_out, d_accumulator, accumulator_size * sizeof(int), hipMemcpyDeviceToHost));

    // 释放设备内存
    CHECK_CUDA_ERROR(hipFree(d_edge_image));
    CHECK_CUDA_ERROR(hipFree(d_accumulator));
    CHECK_CUDA_ERROR(hipFree(d_max_value));
    CHECK_CUDA_ERROR(hipFree(d_max_positions));
    CHECK_CUDA_ERROR(hipFree(d_max_count));
}

}
